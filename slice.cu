#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "param.h"
#include "slice.h"

#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/pair.h>
#include <thrust/copy.h>

unsigned long long int h_v[N1];// for print mast be copied from d_v


int Slice::Init(unsigned int k)
{

	hipError_t err1, err = hipGetLastError();

	//	printf("before all error %d , %s \n",err,hipGetErrorString(err));
	//	if (err!=0)exit(0);
	length = k;
	NN = (((k % SIZE_OF_LONG_INT) == 0) ? (k / SIZE_OF_LONG_INT) : (k / SIZE_OF_LONG_INT + 1));
	//    printf("slice.init %u ", NN);
#ifdef ssss
	int* d_i;
	printf("Slice init %d %s\n", err, hipGetErrorString(err));
	err = hipMalloc(&d_i, sizeof(int));

	d_first_non_zero = d_i;
	printf("Slice alloc error %d %s \n", err, hipGetErrorString(err));
#endif
	//	err1 = hipGetLastError();
	//	printf("before alloc error %d , %s \n",err1,hipGetErrorString(err1));
	err = hipMalloc(&d_v, NN * sizeof(unsigned long long int));
	//   printf("Slice alloc error %d , %s ,%p \n",err,hipGetErrorString(err),d_v);
#ifdef ssss
	printf("Slice alloc error %d %s \n", err, hipGetErrorString(err));
#endif
	hipMemset(d_v, 0, NN * sizeof(unsigned long long int));

	//	exit(0);

	return err;
}

__global__ void set_long_values(unsigned long long int* d_v, unsigned long long int num)
{

	d_v[blockIdx.x] = num;
}

//заполнить единичками,
void Slice::SET()
{
	unsigned long long int zero = 0;
	zero = ~zero;
#ifdef ss
	char s[100];
	long_to_binary(zero, s);
	printf("SET %s \n", s);


	hipError_t err = hipGetLastError();
	printf("error before set_lon_values %d \n", err);
	hipError_t err_c = hipMemcpy(h_v, d_v, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	long_to_binary(h_v[0], s);
	printf("h_v[0] %llu err %d %s\n", h_v[0], err_c, s);
	print("q1", 1);
#endif
	set_long_values << <NN, 1 >> > (d_v, zero);
	//    printf("SET: %i->%llu \n",NN,zero);
#ifdef qq
	err_c = hipMemcpy(h_v, d_v, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	long_to_binary(h_v[0], s);
	printf("h_v[0] %llu err %d %s\n", h_v[0], err_c, s);


	print("q2", 1);

	err = hipGetLastError();
	printf("error after set_lon_values %d \n", err);
#endif
}

//заполнить нулями,
void Slice::CLR()
{
	unsigned long long int zero = 0;
	set_long_values << <NN, 1 >> > (d_v, zero);
}

__global__ void set_mask_values(unsigned long long int* d_v, int num)
{
	unsigned long long int zero = 1;
	int num_el = num >> 6; // номер элемента, содержащий переход от 0 к 1;
	int el = num % SIZE_OF_LONG_INT;
	//  printf("%i in %i \n", num,num_el);
	if (blockIdx.x == num_el)
	{
		zero = (el == 0) ? 0 : (zero << (el - 1)) - 1;
		zero = ~zero;
	}
	else
	{
		zero = 0;
		if (blockIdx.x > num_el)
		{
			zero = ~zero;
		}
	}
	d_v[blockIdx.x] = zero;
}

void Slice::MASK(int i)
{
	set_mask_values << <NN, 1 >> > (d_v, i);
}