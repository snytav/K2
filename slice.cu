#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "param.h"
#include "slice.h"

#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/pair.h>
#include <thrust/copy.h>

unsigned long long int h_v[N1];// for print mast be copied from d_v


int Slice::Init(unsigned int k)
{

	hipError_t err1, err = hipGetLastError();

	//	printf("before all error %d , %s \n",err,hipGetErrorString(err));
	//	if (err!=0)exit(0);
	length = k;
	NN = (((k % SIZE_OF_LONG_INT) == 0) ? (k / SIZE_OF_LONG_INT) : (k / SIZE_OF_LONG_INT + 1));
	//    printf("slice.init %u ", NN);
#ifdef ssss
	int* d_i;
	printf("Slice init %d %s\n", err, hipGetErrorString(err));
	err = hipMalloc(&d_i, sizeof(int));

	d_first_non_zero = d_i;
	printf("Slice alloc error %d %s \n", err, hipGetErrorString(err));
#endif
	//	err1 = hipGetLastError();
	//	printf("before alloc error %d , %s \n",err1,hipGetErrorString(err1));
	err = hipMalloc(&d_v, NN * sizeof(unsigned long long int));
	//   printf("Slice alloc error %d , %s ,%p \n",err,hipGetErrorString(err),d_v);
#ifdef ssss
	printf("Slice alloc error %d %s \n", err, hipGetErrorString(err));
#endif
	hipMemset(d_v, 0, NN * sizeof(unsigned long long int));

	//	exit(0);

	return err;
}
